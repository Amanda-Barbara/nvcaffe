
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype, typename Mtype>
void CuDNNLRNLayer<Dtype,Mtype>::Forward_gpu(const vector<Blob<Dtype,Mtype>*>& bottom,
    const vector<Blob<Dtype,Mtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();

  CUDNN_CHECK(cudnnLRNCrossChannelForward(
        Caffe::cudnn_handle(), norm_desc_, CUDNN_LRN_CROSS_CHANNEL_DIM1,
        cudnn::dataType<Dtype>::one,
        bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        top_desc_, top_data) );



  const Dtype* b_data = bottom[0]->cpu_data();
  const Dtype* t_data = top[0]->cpu_data();
  for (int i = 0; i < bottom[0]->count(); ++i) {
    printf("%g ", (float) b_data[i]);
  }
  printf("\n===================================\n");
  for (int i = 0; i < top[0]->count(); ++i) {
    printf("%g ", (float) t_data[i]);
  }
  printf("\n===================================\n");


}

template <typename Dtype, typename Mtype>
void CuDNNLRNLayer<Dtype,Mtype>::Backward_gpu(const vector<Blob<Dtype,Mtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype,Mtype>*>& bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();

  CUDNN_CHECK(cudnnLRNCrossChannelBackward(
        Caffe::cudnn_handle(), norm_desc_, CUDNN_LRN_CROSS_CHANNEL_DIM1,
        cudnn::dataType<Dtype>::one,
        top_desc_, top_data,
        top_desc_, top_diff,
        bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        bottom_desc_, bottom_diff) );
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNLRNLayer);

}  // namespace caffe
#endif

